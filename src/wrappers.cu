#include <hip/hip_runtime.h>
#include "kernels.cu"
#include <stdio.h>

int forward_step_wrap(float *weight, float *bias, float *a, float *a_new, unsigned int columns, unsigned int rows) {
	float *weight_D, *bias_D, *a_D, *res1_D;
	if (hipMalloc((void **)&weight_D, columns*rows*sizeof(float)) != hipSuccess) {
		return -1;
	}
	if (hipMalloc((void **)&bias_D, rows*sizeof(float)) != hipSuccess) {
		return -2;
	}
	if (hipMalloc((void **)&a_D, ((columns>rows)*columns + (rows>=columns)*rows)*sizeof(float)) != hipSuccess) {
		return -3;
	}
	if (hipMalloc((void **)&res1_D, 64*rows*sizeof(float)) != hipSuccess) {
		return -4;
	}
	
	dim3 block(32, 32, 1);
	dim3 grid(8, 8, rows);
	hipMemcpy(weight_D, weight, columns*rows*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(a_D, a, columns*sizeof(float), hipMemcpyHostToDevice);
	forward_step1<<<grid, block>>>(weight_D, a_D, res1_D, columns);
	hipMemcpy(bias_D, bias, rows*sizeof(float), hipMemcpyHostToDevice);
	block.x = 8;
	block.y = 8;
	grid.x = 1;
	grid.y = 1;
	forward_step2<<<grid, block>>>(res1_D, bias_D, a_D);
	hipMemcpy(a_new, a_D, rows*sizeof(float), hipMemcpyDeviceToHost);
	
	hipFree(weight_D);
	hipFree(bias_D);
	hipFree(a_D);
	hipFree(res1_D);
	
	return 0;
}

int output_error_wrap(float *aL, float *y, float *deltaL, unsigned int output_size) {
	
	float *aL_D, *y_D, *deltaL_D;
	if (hipMalloc((void **)&aL_D, output_size*sizeof(float)) != hipSuccess) {
		return -1;
	}
	if (hipMalloc((void **)&y_D, output_size*sizeof(float)) != hipSuccess) {
		return -2;
	}
	if (hipMalloc((void **)&deltaL_D, output_size*sizeof(float)) != hipSuccess) {
		return -3;
	}
	
	dim3 block(1, 1, 1);
	dim3 grid(1, 1, output_size);
	hipMemcpy(aL_D, aL, output_size*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(y_D, y, output_size*sizeof(float), hipMemcpyHostToDevice);
	output_error<<<grid, block>>>(aL_D, y_D, deltaL_D);
	hipMemcpy(deltaL, deltaL_D, output_size*sizeof(float), hipMemcpyDeviceToHost);
	
	hipFree(aL_D);
	hipFree(y_D);
	hipFree(deltaL_D);
	
	return 0;
}

int backward_step_wrap(float *weight, float *a, float *delta, float *delta_new, unsigned int columns, unsigned int rows) {
	float *weight_D, *a_D, *delta_D, *res1_D;
	if (hipMalloc((void **)&weight_D, columns*rows*sizeof(float)) != hipSuccess) {
		return -1;
	}
	if (hipMalloc((void **)&a_D, columns*sizeof(float)) != hipSuccess) {
		return -2;
	}
	if (hipMalloc((void **)&delta_D, ((columns>rows)*columns + (rows>=columns)*rows)*sizeof(float)) != hipSuccess) {
		return -3;
	}
	if (hipMalloc((void **)&res1_D, 64*columns*sizeof(float)) != hipSuccess) {
		return -4;
	}
	
	dim3 block(32, 32, 1);
	dim3 grid(8, 8, columns);
	
	hipMemcpy(weight_D, weight, columns*rows*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(delta_D, delta, rows*sizeof(float), hipMemcpyHostToDevice);
	backward_step1<<<grid, block>>>(weight_D, delta_D, res1_D, columns, rows);
	hipMemcpy(a_D, a, columns*sizeof(float), hipMemcpyHostToDevice);
	
	block.x = 8;
	block.y = 8;
	grid.x = 1;
	grid.y = 1;
	
	backward_step2<<<grid, block>>>(res1_D, a_D, delta_D);
	hipMemcpy(delta_new, delta_D, columns*sizeof(float), hipMemcpyDeviceToHost);
	
	hipFree(weight_D);
	hipFree(a_D);
	hipFree(delta_D);
	hipFree(res1_D);
	
	return 0;
	
}

//changed needs to be tested
int sum_weights_wrap_test(float *weightGs, float *result1, unsigned int size, unsigned int samples) {
	float *weightGs_D, *result_D;
	unsigned int gridx = samples/1024+1;

	if (hipMalloc((void **)&weightGs_D, size*samples*sizeof(float)) != hipSuccess) {
		return -1;
	}
	if (hipMalloc((void **)&result_D, size*gridx*sizeof(float)) != hipSuccess) {
		return -2;
	}

	dim3 block(32, 32, 1);
	dim3 grid(gridx, size, 1);
	hipMemcpy(weightGs_D, weightGs, size*samples*sizeof(float), hipMemcpyHostToDevice);
	
	sum_of_1024<<<grid, block>>>(weightGs_D, result_D, size, samples);
	grid.x = 1;
	sum_of_1024<<<grid, block>>>(result_D, result_D, size, gridx);
	hipMemcpy(result1, result_D, size*gridx*sizeof(float), hipMemcpyDeviceToHost);
	
	hipFree(weightGs_D);
	hipFree(result_D);
	return 0;
}

int gradient_descent_wrap(float *w_or_b, float *wG_or_bG, unsigned int columns, unsigned int rows, unsigned int samples, float heta) {
	float *w_or_b_D, *wG_or_bG_D;
	unsigned int size = columns*rows;
	if (hipMalloc((void **)&w_or_b_D, size*sizeof(float)) != hipSuccess) {
		return -1;
	}
	if (hipMalloc((void **)&wG_or_bG_D, size*sizeof(float)) != hipSuccess) {
		return -2;
	}

	dim3 block(32, 32, 1);
	dim3 grid(columns/32+1, rows/32+1, 1);

	hipMemcpy(w_or_b_D, w_or_b, size*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(wG_or_bG_D, wG_or_bG, size*sizeof(float), hipMemcpyHostToDevice);
	grad_desc<<<grid, block>>>(w_or_b_D, wG_or_bG_D, size, samples, heta);
	hipMemcpy(w_or_b, w_or_b_D, size*sizeof(float), hipMemcpyDeviceToHost);

	hipFree(w_or_b_D);
	hipFree(wG_or_bG_D);
	return 0;
}

int weight_gradient_wrap(float *a, float *delta, float *weightG, /*size of a*/unsigned int columns, /*size of delta*/unsigned int rows) {
	
	float *a_D, *delta_D, *weightG_D;
	if (hipMalloc((void **)&a_D, columns*sizeof(float)) != hipSuccess) {
		return -1;
	}
	if (hipMalloc((void **)&delta_D, rows*sizeof(float)) != hipSuccess) {
		return -2;
	}
	if (hipMalloc((void **)&weightG_D, columns*rows*sizeof(float)) != hipSuccess) {
		return -3;
	}

	dim3 block(32, 32, 1);
	dim3 grid(columns/32+1, rows/32+1, 1);

	hipMemcpy(a_D, a, columns*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(delta_D, delta, rows*sizeof(float), hipMemcpyHostToDevice);

	weight_gradient<<<grid, block>>>(a_D, delta_D, weightG_D, columns, rows);

	hipError_t error = hipGetLastError();
	if (error != hipSuccess) {
		printf("Error: %s", hipGetErrorString(error));
	}
	hipMemcpy(weightG, weightG_D, columns*rows*sizeof(float), hipMemcpyDeviceToHost);

	hipFree(a_D);
	hipFree(delta_D);
	hipFree(weightG_D);
	return 0;

}